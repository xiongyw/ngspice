#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014, NVIDIA Corporation. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, 
 * are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, 
 *    this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice, 
 *    this list of conditions and the following disclaimer in the documentation and/or 
 *    other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors may be used to 
 *    endorse or promote products derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, 
 * INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, 
 * OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; 
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, 
 * STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "ngspice/ngspice.h"
#include "ngspice/config.h"
#include "resdefs.h"

extern "C"
__global__ void cuRESload_kernel (RESparamGPUstruct, double *, int, int *, double *) ;

extern "C"
int
cuRESload
(
GENmodel *inModel, CKTcircuit *ckt
)
{
    RESmodel *model = (RESmodel *)inModel ;
    int thread_x, thread_y, block_x ;

    hipError_t status ;

#ifdef STEPDEBUG
    SPICE_debug(("entering...\n"));
#endif

    /*  loop through all the resistor models */
    for ( ; model != NULL ; model = model->RESnextModel)
    {
#ifdef STEPDEBUG
        SPICE_debug(("  RESmodName=%s\n", model->RESmodName));
#endif
        /* Determining how many blocks should exist in the kernel */
        thread_x = 1 ;
        thread_y = 256 ;
        if (model->n_instances % thread_y != 0)
            block_x = (int)((model->n_instances + thread_y - 1) / thread_y) ;
        else
            block_x = model->n_instances / thread_y ;

        dim3 thread (thread_x, thread_y) ;

        /* Kernel launch */
        status = hipGetLastError () ; // clear error status
        
#ifdef STEPDEBUG
        SPICE_debug(("  calling cuRESload_kernel() for model %s: block=%d, thread=(%d,%d)\n", model->RESmodName, block_x, thread_x, thread_y));
#endif
        cuRESload_kernel <<< block_x, thread >>> (model->RESparamGPU, ckt->d_CKTrhsOld, model->n_instances,
                                                  model->d_PositionVector, ckt->d_CKTloadOutput) ;

        hipDeviceSynchronize () ;

        status = hipGetLastError () ; // check for launch error
        if (status != hipSuccess)
        {
            fprintf (stderr, "Kernel launch failure in the Resistor Model\n\n") ;
            return (E_NOMEM) ;
        }
    }

    return (OK) ;
}

extern "C"
__global__
void
cuRESload_kernel
(
RESparamGPUstruct RESentry, double *CKTrhsOld, int n_instances, int *d_PositionVector, double * d_CKTloadOutput
)
{
    double m, difference, factor ;

    int instance_ID ;

    instance_ID = threadIdx.y + blockDim.y * blockIdx.x ;
    if (instance_ID < n_instances)
    {
        if (threadIdx.x == 0)
        {
            if (!(RESentry.d_REStc1GivenArray [instance_ID]))
                RESentry.d_REStc1Array [instance_ID] = 0.0 ;
            
            if (!(RESentry.d_REStc2GivenArray [instance_ID]))
                RESentry.d_REStc2Array [instance_ID] = 0.0 ;
            
            if (!(RESentry.d_RESmGivenArray [instance_ID]))
                RESentry.d_RESmArray [instance_ID] = 1.0 ;

            RESentry.d_REScurrentArray [instance_ID] = (CKTrhsOld [RESentry.d_RESposNodeArray [instance_ID]] -
                                                    CKTrhsOld [RESentry.d_RESnegNodeArray [instance_ID]]) *
                                                    RESentry.d_RESconductArray [instance_ID] ;
            
            difference = (RESentry.d_REStempArray [instance_ID] + RESentry.d_RESdtempArray [instance_ID]) - 300.15 ;
            factor = 1.0 + (RESentry.d_REStc1Array [instance_ID]) * difference +
                     (RESentry.d_REStc2Array [instance_ID]) * difference * difference ;
            
            m = (RESentry.d_RESmArray [instance_ID]) / factor ;
            
            d_CKTloadOutput [d_PositionVector [instance_ID]] = m * RESentry.d_RESconductArray [instance_ID] ;
        }
    }

    return ;
}
